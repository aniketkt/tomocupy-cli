#include "hip/hip_runtime.h"
#include "cfunc_fourierrec.cuh"
#include "kernels_fourierrec.cuh"
#include<stdio.h>
cfunc_fourierrec::cfunc_fourierrec(size_t ntheta, size_t pnz, size_t n, size_t theta_)
    : ntheta(ntheta), pnz(pnz), n(n) {
    float eps = 1e-2;
    mu = -log(eps) / (2 * n * n);
    ne = pow(2,ceil(log2(3*n/2)));
    m = ceil(2 * n * 1 / PI * sqrt(-mu * log(eps) + (mu * n) * (mu * n) / 4));    
    hipMalloc((void **)&fde,
            (2 * n + 2 * m) * (2 * n + 2 * m) * pnz * sizeof(half2));
    hipMalloc((void **)&ge,
            (ne/2+1) * ntheta * 2 * pnz * sizeof(half2));
    hipMalloc((void **)&x, n * ntheta * sizeof(float));
    hipMalloc((void **)&y, n * ntheta * sizeof(float));
    
    
    long long ffts[] = {2*n,2*n};
	  long long idist = (2 * n + 2 * m) * (2 * n + 2 * m);long long odist = (2 * n + 2 * m) * (2 * n + 2 * m);
    long long inembed[] = {2 * n + 2 * m, 2 * n + 2 * m};long long onembed[] = {2 * n + 2 * m, 2 * n + 2 * m};
    size_t workSize = 0;

    hipfftCreate(&plan2d);
    hipfftXtMakePlanMany(plan2d, 
        2, ffts, 
        inembed, 1, idist, HIP_C_16F, 
        onembed, 1, odist, HIP_C_16F, 
        pnz, &workSize, HIP_C_16F);    
    // fft 1d
    hipfftCreate(&plan1d);
    ffts[0] = n;
    idist = n;
    odist = n;
    inembed[0] = n;
    onembed[0] = n;
    hipfftXtMakePlanMany(plan1d, 
        1, ffts, 
        inembed, 1, idist, HIP_C_16F, 
        onembed, 1, odist, HIP_C_16F, 
        ntheta*pnz, &workSize, HIP_C_16F);                   

    //fft filter R<->C
    hipfftCreate(&plan_filter_fwd);
    hipfftCreate(&plan_filter_inv);
    
    ffts[0] = ne;
	  idist = ne;odist = ne/2+1;
    inembed[0] = ne;onembed[0] = ne/2+1;
    hipfftXtMakePlanMany(plan_filter_fwd, 
        1, ffts, 
        inembed, 1, idist, HIP_R_16F, 
        onembed, 1, odist, HIP_C_16F, 
        2*ntheta*pnz, &workSize, HIP_C_16F);      
    hipfftXtMakePlanMany(plan_filter_inv, 
        1, ffts, 
        onembed, 1, odist, HIP_C_16F, 
        inembed, 1, idist, HIP_R_16F, 
        2*ntheta*pnz, &workSize, HIP_C_16F);
    
    theta = (float*)theta_;
  }


// destructor, memory deallocation
cfunc_fourierrec::~cfunc_fourierrec() { free(); }

void cfunc_fourierrec::free() {
  if (!is_free) {
    hipFree(fde);
    hipFree(ge);
    hipFree(x);
    hipFree(y);
    hipfftDestroy(plan2d);
    hipfftDestroy(plan1d);
    hipfftDestroy(plan_filter_fwd);
    hipfftDestroy(plan_filter_inv);
    is_free = true;   
  }
}

void cfunc_fourierrec::backprojection(size_t f_, size_t g_, size_t stream_) {
    half2* g = (half2 *)g_;    
    half2* f = (half2 *)f_;
    hipStream_t stream = (hipStream_t)stream_;    
    hipfftSetStream(plan1d, stream);
    hipfftSetStream(plan2d, stream);    

    // set thread block, grid sizes will be computed before cuda kernel execution
    dim3 dimBlock(32,32,1);    
    dim3 GS2d0,GS3d0,GS3d1,GS3d2,GS3d3;  
    GS2d0 = dim3(ceil(n / 32.0), ceil(ntheta / 32.0));
    GS3d0 = dim3(ceil(n / 32.0), ceil(n / 32.0),pnz);
    GS3d1 = dim3(ceil(2 * n / 32.0), ceil(2 * n / 32.0),pnz);
    GS3d2 = dim3(ceil((2 * n + 2 * m) / 32.0),ceil((2 * n + 2 * m) / 32.0), pnz);
    GS3d3 = dim3(ceil(n / 32.0), ceil(ntheta / 32.0),pnz);
   
    
    hipMemsetAsync(fde, 0, (2 * n + 2 * m) * (2 * n + 2 * m) * pnz * sizeof(half2),stream);

    takexy <<<GS2d0, dimBlock, 0, stream>>> (x, y, theta, n, ntheta);


    mulc <<<GS3d3, dimBlock, 0, stream>>> (g, 1/(float)n, n, ntheta, pnz);
    ifftshiftc <<<GS3d3, dimBlock, 0, stream>>> (g, n, ntheta, pnz);
    hipfftXtExec(plan1d, g, g, HIPFFT_FORWARD);
    ifftshiftc <<<GS3d3, dimBlock, 0, stream>>> (g, n, ntheta, pnz);    
    
    gather <<<GS3d3, dimBlock, 0, stream>>> (g, fde, x, y, m, mu, n, ntheta, pnz);    
    wrap <<<GS3d2, dimBlock, 0, stream>>> (fde, n, pnz, m);
    
    fftshiftc <<<GS3d2, dimBlock, 0, stream>>> (fde, 2 * n + 2 * m, pnz);
    hipfftXtExec(plan2d, &fde[m + m * (2 * n + 2 * m)],
               &fde[m + m * (2 * n + 2 * m)], HIPFFT_BACKWARD);
    fftshiftc <<<GS3d2, dimBlock, 0, stream>>> (fde, 2 * n + 2 * m, pnz);
    
    divphi <<<GS3d0, dimBlock, 0, stream>>> (fde, f, mu, n, pnz, ntheta, m);    
    circ <<<GS3d0, dimBlock,0,stream>>> (f, 1.0f / n, n, pnz);    
}

void cfunc_fourierrec::filter(size_t g_, size_t w_, size_t stream_) {
    half* g = (half *)g_;    
    half2* w = (half2 *)w_;
    hipStream_t stream = (hipStream_t)stream_;    
    hipfftSetStream(plan_filter_fwd, stream);
    hipfftSetStream(plan_filter_inv, stream);    
    dim3 dimBlock(32,32,1);        
    dim3 GS3d1 = dim3(ceil(ne/32.0), ceil(ntheta / 32.0),2*pnz);
    dim3 GS3d2 = dim3(ceil((ne/2+1)/32.0), ceil(ntheta / 32.0),2*pnz);
    hipfftXtExec(plan_filter_fwd, g, ge, HIPFFT_FORWARD);
    mulw <<<GS3d2, dimBlock, 0, stream>>> (ge, w, ne/2+1, ntheta, 2*pnz);
    hipfftXtExec(plan_filter_inv, ge, g, HIPFFT_BACKWARD);
}
